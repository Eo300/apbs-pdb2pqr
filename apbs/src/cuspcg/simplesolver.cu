#include <iostream>
#include "cuspcg_cuda.h"
#include <cusp/coo_matrix.h>
#include <cusp/print.h>
#include <cusp/krylov/gs.h>
#include <cusp/multiply.h>


template< typename V >
void 
buildCOOMatrix( CUSP_matrixtype &A, int nx, // A is a nx*nx array of Vs
	V *vA )
{
	int entry = 0;
	CUSP_hostmatrixtype hmtx; // host matrix
	for(int i = 0; i < nx; i++ )
	{
		for(int j = 0; j < nx; j++)
		{
			int idx = i * nx + j;
			if( vA[idx] != 0.0 )
			{
				hmtx.row_indices[entry] = i;
				hmtx.column_indices[entry] = j;
				hmtx.values[entry] = vA[idx];
				entry++;
			}
		}
	}
	A = hmtx;
}

template< typename V >
void 
buildCOOMatrix( CUSP_matrixtype &A, int numNonzeroes, // A is a nx*nx array of Vs
	V *vals, int *rows, int *cols )
{
	CUSP_hostmatrixtype hmtx; // host matrix
	for(int i = 0; i < numNonzeroes; i++ )
	{
		hmtx.row_indices[i] = rows[i];
		hmtx.column_indices[i] = cols[i];
		hmtx.values[i] = vals[i];
	}
	A = hmtx;
}

void* 
newCuspCOOMatrix(int rank, int numNonzeroes)
{
	CUSP_matrixtype *ret = new CUSP_matrixtype(rank,rank,numNonzeroes);

	return (void*)ret;
}

void 
deleteCuspCOOMatrix( void *m )
{
	CUSP_matrixtype *hm = static_cast<CUSP_matrixtype*>(m);
	delete hm;
}

void 
deleteCuspCOOHostMatrix( void *m )
{
	CUSP_hostmatrixtype *hm = static_cast<CUSP_hostmatrixtype*>(m);
	delete hm;
}

void* newCuspCOOHostMatrix(int rank, int numNonzeroes)
{
	CUSP_hostmatrixtype *ret = new CUSP_hostmatrixtype(rank,rank,numNonzeroes);

	return (void*)ret;
}

double 
getCuspCOOMatrixVal( void* matrixptr, int idx)
{
	CUSP_matrixtype* matrix = (CUSP_matrixtype*)matrixptr;
	int row = matrix->row_indices[idx] , col = matrix->column_indices[idx];
	double val = matrix->values[idx];
	std::cout << "(" << row << ", " << col << ") = " << val << std::endl;
	return val;
}

#if 0
void
setCuspCOOMatrixVal( void* matrixptr, int idx, int row, int col, double val )
{
	CUSP_matrixtype* matrix = (CUSP_matrixtype*)matrixptr;

	std::cout << "Setting matrix[" << row << "][" << col << "] = " << val << std::endl;
	
	matrix->row_indices[idx] = row;
	matrix->column_indices[idx] = col;
	matrix->values[idx] = (ValueType)val;
//printf("Set (%d,%d) <- %.3f.  idx is %d\n", row,col,val,idx);
}
#endif


void
setCuspCOOHostMatrixVal( void* matrixptr, int idx, int row, int col, double val )
{
	CUSP_hostmatrixtype* matrix = (CUSP_hostmatrixtype*)matrixptr;

	
	matrix->row_indices[idx] = row;
	matrix->column_indices[idx] = col;
	matrix->values[idx] = (ValueType)val;
//printf("Set (%d,%d) <- %.3f.  idx is %d\n", row,col,val,idx);
}


void*
newCUSParray(int numElems)
{
	return new CUSP_arraytype(numElems);
}

void
deleteCUSParray( void* a )
{
	CUSP_arraytype* ca = static_cast<CUSP_arraytype*>(a);
	delete ca;
}

void*
newCUSPhostarray(int numElems)
{
	return new CUSP_hostarraytype(numElems);
}

void deleteCUSPhostarray( void* a )
{
	CUSP_hostarraytype* ca = static_cast<CUSP_hostarraytype*>(a);
	delete ca;
}


void 
copyHostToDeviceMatrix( void* hmtx, void* dmtx )
{
	CUSP_hostmatrixtype *H = (CUSP_hostmatrixtype*)hmtx;
	CUSP_matrixtype *D = (CUSP_matrixtype*)dmtx;
	*D = *H;
}

void
setCUSParrayVal( void* arrayptr, int idx, double val )
{
	CUSP_arraytype *array = ((CUSP_arraytype*)arrayptr);
	array->operator[](idx) = val;
	
}

void
setCUSPhostarrayVal( void* arrayptr, int idx, double val )
{
	CUSP_hostarraytype *array = ((CUSP_hostarraytype*)arrayptr);
	array->operator[](idx) = val;
	
}

void 
copyArrayh2d( void* host, void* dev )
{
	CUSP_arraytype  *Dev = (CUSP_arraytype*) dev ; 
	CUSP_hostarraytype *Host = (CUSP_hostarraytype*) host;
	
	*Dev = *Host;
}

double
getCUSParrayVal( void* arrayptr, int idx)
{
	//CUSP_arraytype array = *((CUSP_arraytype*)arrayptr);
	//return array[idx];
	CUSP_arraytype *array = ((CUSP_arraytype*)arrayptr);
	return array->operator[](idx);
}

template< typename V >
void
buildCUSPArray( CUSP_arraytype &b, int nx, V *vB )
{
	for(int i = 0; i < nx; i++)
	{
		b[i] = vB[i];
	}
}

void
unpackCUSPArray( CUSP_arraytype *x, int nx, double *vx )
{
	CUSP_hostarraytype hx = *x;
	for(int i = 0; i < nx; i++)
	{
//		vx[i] = x->operator[](i);
		vx[i] = hx[i];
	}
}

/**
* Call the CUSP GS solver on a sparse matrix in COO form.  
* 
* @param numvals  The number of elements in the parallel arrays vals, cols,and rows.
* Also equals the number of nonzero elements in the sparse matrix.
*
* @param vals     Vals, cols, and rows are a set of parallel arrays.  They must hold the
* same number of elements, otherwise behavior is undefined.   Vals holds the nonzero
* elements of the sparse matrix.
*
* @param cols     Vals, cols, and rows are a set of parallel arrays.  They must hold the
* same number of elements, otherwise behavior is undefined.  Cols holds the column number
* of the corresponding element
*
* @param rows     Vals, cols, and rows are a set of parallel arrays.  They must hold the
* same number of elements, otherwise behavior is undefined.   Rows holds the row number of the
* corresponding element.
*/
void CUSP_GS( int* arows,
	      int* acols,
	      double* avals,
	      int numNonzeroes,
	      int xrank,
	      double *_x,
	      double *_b )
{
    // allocate device memory for CSR format
    int* device_I; hipMalloc(&device_I, numNonzeroes * sizeof(int));
    int* device_J; hipMalloc(&device_J, numNonzeroes * sizeof(int));
    double* device_V; hipMalloc(&device_V, numNonzeroes * sizeof(double));
    // allocate device memory for x and y arrays
    double * device_x; hipMalloc(&device_x, xrank * sizeof(double));
    double * device_b; hipMalloc(&device_b, xrank * sizeof(double));

  // copy raw data from host to device
  hipMemcpy(device_I, arows, numNonzeroes * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(device_J, acols, numNonzeroes * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(device_V, avals, numNonzeroes * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(device_x, _x, xrank * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(device_b, _b, xrank * sizeof(double), hipMemcpyHostToDevice);

  
  // *NOTE* raw pointers must be wrapped with thrust::device_ptr!
  thrust::device_ptr<int> wrapped_device_I(device_I);
  thrust::device_ptr<int> wrapped_device_J(device_J);
  thrust::device_ptr<double> wrapped_device_V(device_V);
  thrust::device_ptr<double> wrapped_device_x(device_x);
  thrust::device_ptr<double> wrapped_device_b(device_b);
 
   // use array1d_view to wrap the individual arrays
  typedef typename cusp::array1d_view< thrust::device_ptr<int> > DeviceIndexArrayView;
  typedef typename cusp::array1d_view< thrust::device_ptr<double> > DeviceValueArrayView;

  DeviceIndexArrayView row_indices (wrapped_device_I, wrapped_device_I + numNonzeroes);
  DeviceIndexArrayView column_indices(wrapped_device_J, wrapped_device_J + numNonzeroes);
  DeviceValueArrayView values (wrapped_device_V, wrapped_device_V + numNonzeroes);
  DeviceValueArrayView x (wrapped_device_x, wrapped_device_x + xrank);
  DeviceValueArrayView b (wrapped_device_b, wrapped_device_b + xrank);


  // combine the three array1d_views into a coo_matrix_view
  typedef cusp::coo_matrix_view<DeviceIndexArrayView,
                                DeviceIndexArrayView,
                                DeviceValueArrayView> DeviceView;

  // construct a coo_matrix_view from the array1d_views
  DeviceView A(xrank, xrank, numNonzeroes, row_indices, column_indices, values);
    
  // set stopping criteria: iteration_limit = 100, relative_tolerance = 1e-5
  cusp::verbose_monitor<double> monitor(b, 100, 1e-3);


  // solve the linear system A * x = b with the Gauss-Seidel method
  cusp::krylov::gs(A, wrapped_device_x, wrapped_device_b, xrank, monitor);

  // copy the solution back to the host
  hipMemcpy(_x, device_x, xrank * sizeof(double), hipMemcpyDeviceToHost);

  // free device arrays
  hipFree(device_I);
  hipFree(device_J);
  hipFree(device_V);
  hipFree(device_x);
  hipFree(device_b);

}




/**
* Call the CUSP CG solver on a sparse matrix in COO form.  
* 
* @param numvals  The number of elements in the parallel arrays vals, cols,and rows.
* Also equals the number of nonzero elements in the sparse matrix.
*
* @param vals     Vals, cols, and rows are a set of parallel arrays.  They must hold the
* same number of elements, otherwise behavior is undefined.   Vals holds the nonzero
* elements of the sparse matrix.
*
* @param cols     Vals, cols, and rows are a set of parallel arrays.  They must hold the
* same number of elements, otherwise behavior is undefined.  Cols holds the column number
* of the corresponding element
*
* @param rows     Vals, cols, and rows are a set of parallel arrays.  They must hold the
* same number of elements, otherwise behavior is undefined.   Rows holds the row number of the
* corresponding element.
*/
void CUSP_CG( void* p_cuspA,
	      void* p_cuspx,
	      void* p_cuspb,
	      int nx,
	      double *vals_x )
{

/*	CUSP_matrixtype cuspA( nx, nx, numvals_A );
	CUSP_arraytype cuspx( nx, 0 );
	CUSP_arraytype cuspb( nx, 1 );
	buildCOOMatrix( cuspA, numvals_A, vals_A, rows_A, cols_A );
	buildCUSPArray( cuspx, nx, vals_x );
	buildCUSPArray( cuspb, nx, vals_b );
*/
	CUSP_matrixtype *cuspA = (CUSP_matrixtype*)p_cuspA;
	CUSP_arraytype *cuspx = (CUSP_arraytype*)p_cuspx;
	CUSP_arraytype *cuspb = (CUSP_arraytype*)p_cuspb;

	_CUSP_CG( *cuspA, *cuspx, *cuspb );
	unpackCUSPArray( cuspx, nx, vals_x );
}
	

#if 0
void CUSP_CG( float* A, 
		float *x,
		int nx,  //  A is a nx * nx array of ints
		float* b	//  b is a 'nx' element array of ValueTypes
				// We solve for Ax=b
		)
{
	int numNonzeroes = 0; // count # of nonzeroes in A
	int nxnx = nx * nx;
	for(int i = 0; i < nxnx; i++ )
	{
		if( A[i] != 0.0 ) numNonzeroes++;
	}
	CUSP_matrixtype cuspA( nx, nx, numNonzeroes );
	CUSP_arraytype cuspx( nx, 0 );
	CUSP_arraytype cuspb( nx, 1 );
	buildCOOMatrix( cuspA, nx, A );
	buildCUSPArray( cuspx, nx, x );
	buildCUSPArray( cuspb, nx, b );

//	cusp::multiply( cuspA, cuspx, cuspb );
	

	_CUSP_CG( cuspA, cuspx, cuspb );
	unpackCUSPArray( cuspx, nx, x );

}
#endif

void 
setCuspCOOHostMatrixValues7x( void* vmtx, // CUSP_hostmatrixtype cast to void*
			      int* rows,
			      int* cols,
			      double* vals, // 
			      int numelems ) // number of elements in vals, rows, and cols (these are parallel arrays)
{
	CUSP_hostmatrixtype* matrix = (CUSP_hostmatrixtype*)vmtx;
	for(int idx = 0; idx < numelems; idx++)
	{
		matrix->row_indices[idx] = rows[idx];
		matrix->column_indices[idx] = cols[idx];
		matrix->values[idx] = vals[idx];
	}
}
			

void _CUSP_CG( CUSP_matrixtype &A, CUSP_arraytype &x, CUSP_arraytype &b )
{
    // set stopping criteria:
    //  iteration_limit    = 100
    //  relative_tolerance = 1e-3
//    cusp::verbose_monitor<ValueType> monitor(b, 1000, 1e-3);
    cusp::default_monitor<ValueType> monitor(b, 1000, 1e-4);
    cusp::identity_operator<ValueType, MemorySpace> M(A.num_rows, A.num_rows);
#if DEBUG
	std::cout << "Trying to solve: " << std::endl;
	cusp::print( A );
std::cout << "this is x: " << std::endl;
cusp::print(x);
#endif

    // solve the linear system A * x = b with the Conjugate Gradient method
    cusp::krylov::cg(A, x, b, monitor, M);
//cusp::multiply(A,x,b);
//    cusp::krylov::cg(A, x, b);
	
}


int solvertest(void)
{
    // create an empty sparse matrix structure (HYB format)
    //cusp::hyb_matrix<int, ValueType, MemorySpace> A;
   CUSP_matrixtype A;

    // create a 2d Poisson problem on a 10x10 mesh
    cusp::gallery::poisson5pt(A, 10, 10);

    // allocate storage for solution (x) and right hand side (b)
    cusp::array1d<ValueType, MemorySpace> x(A.num_rows, 0);
    cusp::array1d<ValueType, MemorySpace> b(A.num_rows, 1);

/*
    // set stopping criteria:
    //  iteration_limit    = 100
    //  relative_tolerance = 1e-3
    cusp::verbose_monitor<ValueType> monitor(b, 100, 1e-3);

    // set preconditioner (identity)
    cusp::identity_operator<ValueType, MemorySpace> M(A.num_rows, A.num_rows);

    // solve the linear system A * x = b with the Conjugate Gradient method
    cusp::krylov::cg(A, x, b, monitor, M);

*/
    
    _CUSP_CG( A,x,b );
    return 0;
}

